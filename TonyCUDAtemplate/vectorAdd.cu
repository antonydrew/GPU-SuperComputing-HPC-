#include <stdio.h>

// For the CUDA runtime library/routines (prefixed with "cuda_") - must include this file
#include <hip/hip_runtime.h>

/* CUDA Kernel Device code
 * Computes the vector addition of 10 to each iteration i */
__global__ void kernelTest(int* i, int length){

    unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;

    if(tid < length)
        i[tid] = i[tid] + 10;}

/* This is the main routine which declares and initializes the integer vector, moves it to the device, launches kernel
 * brings the result vector back to host and dumps it on the console. */
int main(){
	
	//declare pointer and allocate memory for host CPU variable - must use MALLOC of CudaHostAlloc here
    int length  = 100;
    int* i = (int*)malloc(length*sizeof(int));

	//fill CPU variable with values from 1 to 100 via loop
    for(int x=0;x<length;x++)
        i[x] = x;

	//declare pointer and allocate memory for device GPU variable denoted with "_d"
    int* i_d;
    hipMalloc((void**)&i_d,length*sizeof(int));

	//copy contents of host CPU variable over to GPU variable on GPU device
    hipMemcpy(i_d, i, length*sizeof(int), hipMemcpyHostToDevice);

	//designate how many threads and blocks to use on GPU for CUDA function call/calculation - this depends on each device
    dim3 threads; threads.x = 256;
    dim3 blocks; blocks.x = (length/threads.x) + 1;

	//call CUDA C function - note triple chevron here - this is CUDA syntax
    kernelTest<<<threads,blocks>>>(i_d,length);
	
	//wait for CUDA C function to finish and then copy results from GPU variable on device back over to CPU variable on host
    hipMemcpy(i, i_d, length*sizeof(int), hipMemcpyDeviceToHost);

	//print results of CPU variable to console
    for(int x=0;x<length;x++)
        printf("%d\t",i[x]);

	//free memory for both CPU and GPU variables/pointers
	free (i); hipFree (i_d);

	//reset GPU device
	system("pause");
	hipDeviceReset();  }
	