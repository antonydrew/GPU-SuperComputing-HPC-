/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */



#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void kernelTest(int* i, int length){

    unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;

    if(tid < length)
        i[tid] = i[tid] + 10;
}

/*
 * This is the main routine which declares and initializes the integer vector, moves it to the device, launches kernel
 * brings the result vector back to host and dumps it on the console.
 */
int main(){

    int length  = 100;
    int* i = (int*)malloc(length*sizeof(int));

    for(int x=0;x<length;x++)
        i[x] = x;

    int* i_d;
    hipMalloc((void**)&i_d,length*sizeof(int));

    hipMemcpy(i_d, i, length*sizeof(int), hipMemcpyHostToDevice);

    dim3 threads; threads.x = 256;
    dim3 blocks; blocks.x = (length/threads.x) + 1;

    kernelTest<<<threads,blocks>>>(i_d,length);
	

    hipMemcpy(i, i_d, length*sizeof(int), hipMemcpyDeviceToHost);

    for(int x=0;x<length;x++)
        printf("%d\t",i[x]);

	system("pause");


}