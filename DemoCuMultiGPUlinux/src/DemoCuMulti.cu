#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <float.h>
#include <time.h>
#include <malloc.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
//#include <sys/io.h>

//#pragma warning( disable : 4996 )

//Declaring macros and constants in pre-processor - STEP A//INSERT NEW STUFF DOWN HERE EACH TIME START SUB-FUNCTION******PRE-PROCESSOR AREA*******//
#define ABS(X) (X>=0?X:-X)
#define MAX(X,Y) (X>=Y?X:Y)
#define MIN(X,Y) (X<=Y?X:Y)
#define SIGN(X) (X>=0?(X==0?0:1):-1)
#define ROUND(X,Y) ((X>=0?(X<<1)+Y:(X<<1)-Y)/(Y<<1))*Y

//Change path below for UNIX "c://usr"
#define PATH "/home/tony/DemoCuMultiGPU/"
#define LOOKBACK 1597		// 1597-987-610-377-144-89 fibos rolling optimization historical period
#define STEP 377			// or 89 fibos step forward in time period for next rolling optimization
#define NUMI 27				//up to 27 number of markets
#define STR 2				//number of streams - use 2 to be safe since most GPU's can handle 2 streams if threads are low on each stream
#define THR 256				//initial threads - must specify this PARAM - can affect OCCUPANCY or %USE OF GPU

long f_line(FILE *f);
FILE * f_openr( char *filer);
FILE * f_openw( char *filer);
void f_close(char *filer,FILE *f);
void snf(double *op, double *lo, double *hi, long end, double *price, double *smooth, double *detrender, double *period, double *qu, double *iu, double *ji, double *jq, double *ib, double *qb, double *sib, double *sqb, double *re, double *im, double *sre, double *sim, double *speriod, double *smperiod, double *qc, double *ic, double *ric, int *intperiod, double *sig, double *nois, double *snr);
void zscore(int lens, double *op, double *sumv, double *varv, long end, float *zscores, double *stdevv, double *m_avev);
void ret(double *p, long end, float *rets);
void sharpe(float *pnl, double *sumi, double *vari, long end, double *stdevi, double *m_avei, double *sharp);
void sharpep(float *pnl, double *sumip, double *varip, int start, int stop, double *stdevip, double *m_aveip, double *sharpp);
__global__ void kernelSim(float *zscores_d,float *rets_d,float *pnl_d,float *pos_d,int start,int stop,float zcut, int lens);

//INT MAIN//INSERT NEW STUFF HERE EACH TIME START SUB-FUNCTION*******MAIN AREA****//Declare each new variable here - initializing and declaring space/memory for return arrays of variables or output we want****STEP B//
int main(int argc, char **argv){


	FILE *recon, *fin, *ferr, *fins,*ferri;
	int *intperiod,*start_h,*stop_h,c, lens=0, combos=0,counters=0,starto=0,tachy=0,lenny=0,gap=0,gap2=0,GPUn=0,gapn=0,gapo=0,dd=0,startg=0,stopg=0;
	long *dt;
	float *zscores_d, *pnl_d, *pos_d, *rets_d,*a_d,*a_h,*pos,*pnl,*zscores,*rets;
	double *op,*hi,*lo,*p,*price, *smooth, *detrender, *period, *qu, *iu, *ji, *jq, *ib, *qb, *sib, *sqb,*re, *im,*sre,*sim,*speriod,*smperiod,*qc,*ic,*ric,*sig,*nois,*snr, *cumpnl, *sharp;
	double *sumi, *vari, *stdevi, *m_avei,*sumv, *varv, *stdevv, *m_avev, *dolls, *cumdolls, *sumip, *varip, *stdevip, *m_aveip, *sharpp;
	int i=0,combo=0,ii=0,zz=0,wins=0,counter=0,start=1,stop=0,*start_d,*stop_d,startf=0,stopf=0,beg=1,high=0,m=0,mm=0,gg=0; char desty[50],dest[50],destr[50],desta[50],tmp[50],strs[50],foldr[50],fnum[50],fnums[50],fnumss[50],dir[50]; int peri[100] = { { 0 } };

	double pp[] = { 42000.00, 42000.00, 50.00, 20.00, 100.00, 100.00, 10.00, 25.00, 5.00, 1000.00, 1000.00, 2000.00, 1000.00, 2500.00, 100.00, 25000.00, 5000.00, 50.00, 100000.00, 125000.00, 125000.00, 125000.00, 62500.00, 50.00, 1000.00, 10000.00, 50.00 };
	char *marks[] = {"RBOB","HO","SP", "ND", "EMD", "TF", "FESX", "FDAX", "NK", "US", "TY", "TU", "FGBL", "ED", "GC", "HG", "SI", "PL", "AD", "EC", "SF", "JY", "BP", "S", "CL", "NG", "C"};
	char *dfiles[] = {PATH"data0.dat",PATH"data1.dat",PATH"data2.dat",PATH"data3.dat",PATH"data4.dat",PATH"data5.dat",PATH"data6.dat",PATH"data7.dat",PATH"data8.dat",PATH"data9.dat",PATH"data10.dat",PATH"data11.dat",PATH"data12.dat",PATH"data13.dat",
		PATH"data14.dat",PATH"data15.dat",PATH"data16.dat",PATH"data17.dat",PATH"data18.dat",PATH"data19.dat",PATH"data20.dat",PATH"data21.dat",PATH"data22.dat",PATH"data23.dat",PATH"data24.dat",PATH"data25.dat",PATH"data26.dat"};

	char sources[60],source[60],line[100]; long end, endf;
	double a[] = { 1.25, 1.50 };		//array holder for parameter combinations later on aka "parameter sweeps" which GPU can greatly speed up// a[] is # standard deviations//
	double b[] = { 21.00, 34.00 };
	//double b[] = { 3.0, 5.0, 8.0, 10.0 };
	double lensa=sizeof(a)/sizeof(double); double maxi=0.00,mat=0.00;
	double lensb=sizeof(b)/sizeof(double);
	double lensc=lensa * lensb;				//number of parameter combinations
	double sharplist[100][7][100] = { { 0 } };
	double table[100][5]= { { 0 } }; double ocum[100][3][100]= { { 0 } };
	int z=0, j=0, winos=0,lensz=0; double sumss[100] = { { 0 } }; double avv=0.00, sharpie=0.00, sharpies=0.00;
	
	//CHECK for USER INPUT larger than MAX NUMI of Markets
	if (NUMI > 27)
    {
        fprintf(stderr, "You entered too many markets! MAX number is 27! Please try again!\n");
        exit(EXIT_FAILURE); }

	// Error code to check return values for CUDA calls - check for num of GPU devices and return error in NA
    	hipError_t err = hipSuccess;
	err=hipGetDeviceCount(&GPUn);

	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to find GPU devices  (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); }

	//loop thru as many GPU devices as there are available - so now we are MULTI-STREAMING and using MULTI-GPU's - this is FULL POWER of GPU computing
for (dd = 0; dd < GPUn; dd++) {

	clock_t ff, ss; float diff=0.00f;
	ff = dd;
	ff = clock();
	strcpy(destr, PATH"recon");		//output RECON directory for each GPU
	strcpy(foldr, ".dat"); 
	sprintf(fnumss, "%d", dd);
	strcat(destr,fnumss);  
	strcat(destr,foldr); 
	recon=f_openw(destr);	

	err=hipSetDevice(dd);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to find GPU device  (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); }

	//divide market data array up into blocks of markets to run on EACH GPU
	gapn = NUMI/GPUn;
	gapo = NUMI/GPUn;
	startg = 1 + (dd*gapn);

	//check for odd number of markets divided by GPUs and append odd number to last iteration of this outer loop
	if (dd == GPUn-1 && NUMI % GPUn != 0) gapn = (((int)((NUMI % GPUn)* GPUn))+gapo)-1;
	if (gapn == 0) gapn = 1;

for (gg = 1; gg <= gapn; gg++) {		// top loop for number of market data files passed thru dfiles[] // must change NUMI in #def as add number of markets // should I use STRUCT instead to store file names?

	
	sprintf(sources,dfiles[(gg-1)+(dd*gapo)]);			//find and open price data files to get lengths for periodicities//
	fins=f_openr(sources);
	endf=f_line(fins);
	endf--;
	peri[gg] = (int)(((endf-LOOKBACK)/STEP)+1); //number of rolling periods in each data set for rolling optimization (aka moving average)//
	f_close(sources,fins);
	sprintf(fnums, "%d", gg);
	strcpy(desta, PATH"OSrunALL");		//output directory for out-of-sample tests for all combined tests per market
	strcpy(foldr, ".dat");
	strcat(desta,"-");
	strcat(desta,fnums);
	strcat(desta,"-");
	strcat(desta,marks[(gg-1)+(dd*gapo)]);
	strcat(desta,foldr);
	ferri=f_openw(desta);

 for (ii = 1; ii <= peri[gg] ; ii++) {	// loop is for periodicity - so 30yrs of price data divided into sub-units for rolling optimization (aka parameter sweeps)
  for (z = 0; z < lensa; z++) {			// 2 nested for loops for parameter sweep or combination of arrays a[] and b[]//
   for (j = 0; j < lensb; j++) {

	lens = (int)(b[j]);
	mm=(lensa*z)+j;
	sprintf(strs, "%d", mm);
	sprintf(fnum, "%d", gg);
	sprintf(tmp, "%d", ii);
	strcpy(desty, PATH"ISout");		//output directory for in-sample tests
	strcpy(foldr, ".dat");
	strcat(desty,strs);
	strcat(desty,"-");
	strcat(desty,fnum);
	strcat(desty,"-");
	strcat(desty,tmp);
	strcat(desty,"-");
	strcat(desty,marks[(gg-1)+(dd*gapo)]);
	strcat(desty,foldr);
	ferr=f_openw(desty);				//find and open output file//
	sprintf(source,dfiles[(gg-1)+(dd*gapo)]);			//find and open price data file//
	fin=f_openr(source);
	end=f_line(fin);
	end--;
	start = beg + (STEP*(ii-1));		//start stop dates for inner loop ii for rolling period optimizations
	stop = LOOKBACK + (STEP*(ii-1));

	dt=(long*) calloc(end+1,sizeof(long));
	op=(double*) calloc(end+1,sizeof(double));
	hi=(double*) calloc(end+1,sizeof(double));
	lo=(double*) calloc(end+1,sizeof(double));
	p=(double*) calloc(end+1,sizeof(double));
	price=(double*) calloc(end+1,sizeof(double));
	smooth=(double*) calloc(end+1,sizeof(double));
	detrender=(double*) calloc(end+1,sizeof(double));
	period=(double*) calloc(end+1,sizeof(double));
	qu=(double*) calloc(end+1,sizeof(double));
	iu=(double*) calloc(end+1,sizeof(double));
	ji=(double*) calloc(end+1,sizeof(double));
	jq=(double*) calloc(end+1,sizeof(double));
	ib=(double*) calloc(end+1,sizeof(double));
	qb=(double*) calloc(end+1,sizeof(double));
	sib=(double*) calloc(end+1,sizeof(double));
	sqb=(double*) calloc(end+1,sizeof(double));
	re=(double*) calloc(end+1,sizeof(double));
	im=(double*) calloc(end+1,sizeof(double));
	sre=(double*) calloc(end+1,sizeof(double));
	sim=(double*) calloc(end+1,sizeof(double));
	speriod=(double*) calloc(end+1,sizeof(double));
	smperiod=(double*) calloc(end+1,sizeof(double));
	qc=(double*) calloc(end+1,sizeof(double));
	ic=(double*) calloc(end+1,sizeof(double));
	ric=(double*) calloc(end+1,sizeof(double));
	intperiod=(int*) calloc(end+1,sizeof(double));
	sig=(double*) calloc(end+1,sizeof(double));
	nois=(double*) calloc(end+1,sizeof(double));
	snr=(double*) calloc(end+1,sizeof(double));


	//** ALLOCATE SPACE FOR MEMORY FOR CUDA-RELATED HOST VARIABLES - USE PINNED/SHARED MEMORY FOR MORE SPEED!!! **

	//rets=(float*) calloc(end+1,sizeof(float));
	hipHostAlloc(&rets, (int)(end)*sizeof(float), hipHostMallocDefault);
	//memset(rets, 0, (int)(end)*sizeof(float));
	dolls=(double*) calloc(end+1,sizeof(double));
	cumdolls=(double*) calloc(end+1,sizeof(double));
	//pos=(float*) calloc(end+1,sizeof(float));
	hipHostAlloc(&pos, (int)(end)*sizeof(float), hipHostMallocDefault);
	//memset(pos, 0, (int)(end)*sizeof(float));
	//pnl=(float*) calloc(end+1,sizeof(float));
	hipHostAlloc(&pnl, (int)(end)*sizeof(float), hipHostMallocDefault);
	//memset(pnl, 0, (int)(end)*sizeof(float));
	cumpnl=(double*) calloc(end+1,sizeof(double));
	//zscores=(float*) calloc(end+1,sizeof(float));
	hipHostAlloc(&zscores, (int)(end)*sizeof(float), hipHostMallocDefault);
	//memset(zscores, 0, (int)(end)*sizeof(float));

	sharpp=(double*) calloc(end+1,sizeof(double));
	sumip=(double*) calloc(end+1,sizeof(double));
	varip=(double*) calloc(end+1,sizeof(double));
	stdevip=(double*) calloc(end+1,sizeof(double));
	m_aveip=(double*) calloc(end+1,sizeof(double));
	sharp=(double*) calloc(end+1,sizeof(double));
	sumi=(double*) calloc(end+1,sizeof(double));
	vari=(double*) calloc(end+1,sizeof(double));
	stdevi=(double*) calloc(end+1,sizeof(double));
	m_avei=(double*) calloc(end+1,sizeof(double));
	sumv=(double*) calloc(end+1,sizeof(double));
	varv=(double*) calloc(end+1,sizeof(double));
	stdevv=(double*) calloc(end+1,sizeof(double));
	m_avev=(double*) calloc(end+1,sizeof(double));


	//** ALLOCATE SPACE FOR MEMORY FOR CUDA-RELATED DEVICE VARIABLES**

	hipMalloc((void**)&zscores_d, (int)(end)*sizeof(float));
	//hipMemset(zscores_d, 0, (int)(end)*sizeof(float));
	hipMalloc((void**)&pos_d, (int)(end)*sizeof(float));
	hipMemset(pos_d, 0, (int)(end)*sizeof(float));
	hipMalloc((void**)&pnl_d, (int)(end)*sizeof(float));
	hipMemset(pnl_d, 0, (int)(end)*sizeof(float));
	hipMalloc((void**)&rets_d, (int)(end)*sizeof(float));
	//hipMemset(rets_d, 0, (int)(end)*sizeof(float));
	/*hipMalloc((void**)&a_d, 1);
	hipMalloc((void**)&start_d, 1);
	hipMalloc((void**)&stop_d, 1);*/



//INSERT NEW STUFF HERE EACH TIME START SUB-FUNCTION*************///CALLING function//STEP C//

	i=0;
	while(fgets(line,100,fin)>0){
		sscanf(line,"%ld %lf %lf %lf %lf",&dt[i],&op[i],&hi[i],&lo[i],&p[i]);i++;}  //scan lines from data file and store in arrays - this is price data here//
	f_close(source,fin);															//close data file

	//Using or CALLING function here//  DO NOT NEED TO DEFINE INPUTS-OUTPUTS here - that is done at bottom down BELOW!!//

	ret(p, end, rets);
	//snf(op, lo, hi, end, price, smooth, detrender, period, qu, iu, ji, jq, ib, qb, sib, sqb, re, im, sre, sim, speriod, smperiod, qc, ic, ric, intperiod, sig, nois, snr);
	zscore(lens, p, sumv, varv, end, zscores, stdevv, m_avev);
	m = (lensa*z)+j;

	//** COPY CUDA VARIABLES FROM CPU (HOST) TO GPU (DEVICE) - USE ASYNC TRANSFER FOR MORE SPEED SO CPU DOES NOT HAVE TO WAIT FOR GPU TO FINISH OPERATION AND CAN PROCEED FURTHER IN THE MAIN PROGRAM**
	hipMemcpyAsync(zscores_d, zscores, (int)(end)*sizeof(float), hipMemcpyHostToDevice,0);
	hipMemcpyAsync(rets_d, rets, (int)(end)*sizeof(float), hipMemcpyHostToDevice,0);

	gap = (stop - start)/STR;
	lenny=stop-start;
	dim3 threads; threads.x = THR;		//use 896 threads as per specific GPU device for higher OCCUPANCY/USE OF CARD - trial-and-error via PROFILING
    dim3 blocks; blocks.x = (lenny/threads.x) + 1;  //max blocks is 112 on GTX 670 device
	//kernelSim<<<threads,blocks>>>(zscores_d,rets_d,pnl_d,pos_d,start,stop,a[z]);

	// allocate and initialize an array of stream handles
    hipStream_t *streams = (hipStream_t *) malloc(STR * sizeof(hipStream_t));

	//** Create Streams for Concurrency or Multi-Streaming - now we will call several KERNELS simultaneously**
	for(int i = 0; i < STR; i++) hipStreamCreate(&(streams[i]));

	//** CALL GPU FUNCTION/KERNEL HERE FOR MODEL PARAMETER SWEEP TO GENERATE IN_SAMPLE RESULTS**THIS IS THREAD REDUCTION DUE TO CONCURRENCY!
    //kernelSim<<<threads,32>>>(zscores_d,rets_d,pnl_d,pos_d,start,stop,(float)(a[z]),lens);
	for (i = 0; i < STR; i++)
		{
			kernelSim<<<32,threads,0,streams[i]>>>(zscores_d,rets_d,pnl_d,pos_d,start+(i*gap),start+((i+1)*gap),(float)(a[z]),lens);
			if (i == STR-1) kernelSim<<<32,threads,0,streams[i]>>>(zscores_d,rets_d,pnl_d,pos_d,start+(i*gap),stop,(float)(a[z]),lens);
		}

	//SYNC up STREAMS before copying back data to CPU
	hipStreamSynchronize(streams[STR-1]);

	//** COPY CUDA VARIABLES/RESULTS FROM GPU (DEVICE) BACK TO CPU (HOST) - MUST WAIT FOR GPU OPERATION/FUNCTION TO FINISH HERE SINCE LOW ASYNC/CONCURRENCY ON NON_TESLA GPU DEVICES**
	hipMemcpy(pos, pos_d, (int)(end)*sizeof(float)/*stop-start*/, hipMemcpyDeviceToHost);
	hipMemcpy(pnl, pnl_d, (int)(end)*sizeof(float), hipMemcpyDeviceToHost);

	//** Destroy Streams for Concurrency or Multi-Streaming - now we will RELEASE resources back to GPU**
	for(int i = 0; i < STR; i++) hipStreamDestroy(streams[i]);

	//for(i=start;i<stop;i++){														//IN-sample rolling optimization for old CPU CODE - NOW WE'RE USING GPU INSTEAD FOR MORE SPEED**
	//
	//	if(zscores[i] > a[z]) pos[i] = 1.00;
	//	if(zscores[i] < -a[z]) pos[i] = -1.00;
	//	pnl[i] = (pos[i] * rets[i]); }


		sharpep(pnl, sumip, varip, start, stop, stdevip, m_aveip, sharpp);
		table[m][0] = m;
		table[m][1] = a[z];
		table[m][2] = b[j];
		table[m][3] = sharpp[stop-1];
		sharpie = sharpie + sharpp[stop-1];//end?
		if (table[m][3] > 0.00) combo = combo + 1;
		counter=counter+1;
		//table[m][4] = cumpnl[stop-1];
		printf("\nIS Test%.0f Market%d-%s Period%d", table[m][0],gg,marks[(gg-1)+(dd*gapo)],ii);
		printf("\nSharpe\t%.2f", table[m][3]);
		printf("\nParam1\t%.2f", table[m][1]);
		printf("\nParam2\t%.0f", table[m][2]);
		//printf("\nCum Ret\t%.2f%%", table[m][4]*100);
		//printf("\nAnn Ret\t%.2f%%", (table[m][4]*100)/(LOOKBACK/260));
		//printf("\nAnn Vol\t%.2f%%", ABS(((table[m][4]*100)/(LOOKBACK/260))/table[m][3]));
		printf("\nNum of Years: %.2f thru %.2f of %.2f total\n", ((((ii-1)*(double)(STEP)))/260),(((double)(LOOKBACK) + (ii*(double)(STEP)))/260)-(double)(STEP)/260,((double)(end)/260));
		fprintf(recon,"\nIS Test%.0f Market%d-%s Period%d", table[m][0],gg,marks[(gg-1)+(dd*gapo)],ii);
		fprintf(recon,"\nSharpe\t%.2f", table[m][3]);
		fprintf(recon,"\nParam1\t%.2f", table[m][1]);
		fprintf(recon,"\nParam2\t%.0f", table[m][2]);
		//fprintf(recon,"\nCum Ret\t%.2f%%", table[m][4]*100);
		//fprintf(recon,"\nAnn Ret\t%.2f%%", (table[m][4]*100)/(LOOKBACK/260));
		//fprintf(recon,"\nAnn Vol\t%.2f%%", ABS(((table[m][4]*100)/(LOOKBACK/260))/table[m][3]));
		fprintf(recon,"\nNum of Years: %.2f thru %.2f of %.2f total\n", ((((ii-1)*(double)(STEP)))/260),(((double)(LOOKBACK) + (ii*(double)(STEP)))/260)-(double)(STEP)/260,((double)(end)/260));

	for(i=start;i<stop;i++) {
		fprintf(ferr,"%ld\t %10.6lf\t %10.3lf\t %10.2lf\t %10.5lf\t %10.5lf\t %10.5lf\n",dt[i],p[i],zscores[i],pos[i],rets[i],pnl[i],sharpp[i]); } //

	f_close(desty,ferr);														//close output file

	for (i = 0; i < lensc; i++)													//find best sharpe ratio from table
            {
                if (table[i][3] > maxi) maxi = table[i][3];
                if (maxi == table[i][3]) high=i;
            }

			sharplist[gg][0][ii] = high;										//row of max sharpe recap
			sharplist[gg][6][ii] = table[high][3];								//max sharpe
			sharplist[gg][1][ii] = table[high][1];								//param 1 recap
			sharplist[gg][2][ii] = table[high][2];								//param 2 recap
			sharplist[gg][3][ii] = table[high][4];								//cum ret recap
			sharplist[gg][4][ii] = table[high][0];								//test number recap
			sharplist[gg][5][ii] = gg;											//market number recap

            maxi=0.00;

//ADD IN EACH POINTER VARIABLE HERE - FREEING UP SPACE IN MEMORY*******STEP D//

	//hipDeviceReset();

	free(hi);free(lo);free(p);free(price);free(smooth);free(detrender);free(period);free(qu);free(iu);free(ji);free(jq);free(ib);free(qb);free(sib);free(re);free(im);free(sre);free(sim);free(speriod);free(smperiod);free(qc);free(ic);free(ric);free(intperiod);free(sig);free(nois);free(snr);
	hipHostFree(rets);hipHostFree(zscores);free(cumpnl);free(op);free(sharp);free(sumi);free(vari);free(stdevi);free(m_avei);
	free(sumv);free(varv);free(stdevv);free(m_avev);free(dolls);free(cumdolls);free(sharpp);free(sumip);free(varip);free(stdevip);free(m_aveip);//free(a_h);free(start_h);free(stop_h);
	hipFree(zscores_d);hipFree(pnl_d);hipFree(pos_d);hipFree(rets_d);hipHostFree(pos);hipHostFree(pnl);
	//hipHostFree(zscores_d);hipHostFree(pnl_d);hipHostFree(pos_d);hipHostFree(rets_d);

		}
	  }

	avv = avv+sharplist[gg][6][ii]/peri[gg];									//avg max sharpe
	if (sharplist[gg][6][ii] > 0.00) wins = wins+ii/peri[gg];					//winning markets
	printf("\n%.2f Max Sharpe of Market%d-%s Period%d is Test %.0f with STD %.2f and MA %.0f\n", sharplist[gg][6][ii],gg,marks[(gg-1)+(dd*gapo)],ii,sharplist[gg][0][ii],sharplist[gg][1][ii],sharplist[gg][2][ii]);
	fprintf(recon,"\n%.2f Max Sharpe of Market%d-%s Period%d is Test %.0f with STD %.2f and MA %.0f\n", sharplist[gg][6][ii],gg,marks[(gg-1)+(dd*gapo)],ii,sharplist[gg][0][ii],sharplist[gg][1][ii],sharplist[gg][2][ii]);


	sprintf(fnum, "%d", gg);
	sprintf(tmp, "%d", ii);
	strcpy(dest, PATH"OSrun");		//output directory for out-of-sample tests
	strcpy(foldr, ".dat");
	strcat(dest,"-");
	strcat(dest,fnum);
	strcat(dest,"-");
	strcat(dest,tmp);
	strcat(dest,"-");
	strcat(dest,marks[(gg-1)+(dd*gapo)]);
	strcat(dest,foldr);
	ferr=f_openw(dest);				//find and open output file//
	sprintf(source,dfiles[(gg-1)+(dd*gapo)]);	//find and open price data file//
	fin=f_openr(source);
	end=f_line(fin);
	end--;


	dt=(long*) calloc(end+1,sizeof(long));
	op=(double*) calloc(end+1,sizeof(double));
	hi=(double*) calloc(end+1,sizeof(double));
	lo=(double*) calloc(end+1,sizeof(double));
	p=(double*) calloc(end+1,sizeof(double));
	price=(double*) calloc(end+1,sizeof(double));
	smooth=(double*) calloc(end+1,sizeof(double));
	detrender=(double*) calloc(end+1,sizeof(double));
	period=(double*) calloc(end+1,sizeof(double));
	qu=(double*) calloc(end+1,sizeof(double));
	iu=(double*) calloc(end+1,sizeof(double));
	ji=(double*) calloc(end+1,sizeof(double));
	jq=(double*) calloc(end+1,sizeof(double));
	ib=(double*) calloc(end+1,sizeof(double));
	qb=(double*) calloc(end+1,sizeof(double));
	sib=(double*) calloc(end+1,sizeof(double));
	sqb=(double*) calloc(end+1,sizeof(double));
	re=(double*) calloc(end+1,sizeof(double));
	im=(double*) calloc(end+1,sizeof(double));
	sre=(double*) calloc(end+1,sizeof(double));
	sim=(double*) calloc(end+1,sizeof(double));
	speriod=(double*) calloc(end+1,sizeof(double));
	smperiod=(double*) calloc(end+1,sizeof(double));
	qc=(double*) calloc(end+1,sizeof(double));
	ic=(double*) calloc(end+1,sizeof(double));
	ric=(double*) calloc(end+1,sizeof(double));
	intperiod=(int*) calloc(end+1,sizeof(double));
	sig=(double*) calloc(end+1,sizeof(double));
	nois=(double*) calloc(end+1,sizeof(double));
	snr=(double*) calloc(end+1,sizeof(double));

	//** ALLOCATE SPACE FOR MEMORY FOR CUDA-RELATED HOST VARIABLES - USE PINNED/SHARED MEMORY FOR MORE SPEED!!! **

	//rets=(float*) calloc(end+1,sizeof(float));
	hipHostAlloc(&rets, (int)(end)*sizeof(float), hipHostMallocDefault);
	//memset(rets, 0, (int)(end)*sizeof(float));
	dolls=(double*) calloc(end+1,sizeof(double));
	cumdolls=(double*) calloc(end+1,sizeof(double));
	//pos=(float*) calloc(end+1,sizeof(float));
	hipHostAlloc(&pos, (int)(end)*sizeof(float), hipHostMallocDefault);
	//memset(pos, 0, (int)(end)*sizeof(float));
	//pnl=(float*) calloc(end+1,sizeof(float));
	hipHostAlloc(&pnl, (int)(end)*sizeof(float), hipHostMallocDefault);
	//memset(pnl, 0, (int)(end)*sizeof(float));
	cumpnl=(double*) calloc(end+1,sizeof(double));
	//zscores=(float*) calloc(end+1,sizeof(float));
	hipHostAlloc(&zscores, (int)(end)*sizeof(float), hipHostMallocDefault);
	//memset(zscores, 0, (int)(end)*sizeof(float));

	sharpp=(double*) calloc(end+1,sizeof(double));
	sumip=(double*) calloc(end+1,sizeof(double));
	varip=(double*) calloc(end+1,sizeof(double));
	stdevip=(double*) calloc(end+1,sizeof(double));
	m_aveip=(double*) calloc(end+1,sizeof(double));
	sharp=(double*) calloc(end+1,sizeof(double));
	sumi=(double*) calloc(end+1,sizeof(double));
	vari=(double*) calloc(end+1,sizeof(double));
	stdevi=(double*) calloc(end+1,sizeof(double));
	m_avei=(double*) calloc(end+1,sizeof(double));
	sumv=(double*) calloc(end+1,sizeof(double));
	varv=(double*) calloc(end+1,sizeof(double));
	stdevv=(double*) calloc(end+1,sizeof(double));
	m_avev=(double*) calloc(end+1,sizeof(double));

	//** ALLOCATE SPACE FOR MEMORY FOR CUDA-RELATED DEVICE VARIABLES**

	hipMalloc((void**)&zscores_d, (int)(end)*sizeof(float));
	//hipMemset(zscores_d, 0, (int)(end)*sizeof(float));
	hipMalloc((void**)&pos_d, (int)(end)*sizeof(float));
	hipMemset(pos_d, 0, (int)(end)*sizeof(float));
	hipMalloc((void**)&pnl_d, (int)(end)*sizeof(float));
	hipMemset(pnl_d, 0, (int)(end)*sizeof(float));
	hipMalloc((void**)&rets_d, (int)(end)*sizeof(float));


	starto = LOOKBACK + (STEP*(ii-1));
	stopf = LOOKBACK + (STEP*(ii-0));
	if(ii>1) tachy = 1;			//use to go back n peroids for max sharpe offset
	if(stopf>=end) stopf = end;

	i=0;
	while(fgets(line,100,fin)>0){
		sscanf(line,"%ld %lf %lf %lf %lf",&dt[i],&op[i],&hi[i],&lo[i],&p[i]);i++;}  //scan lines from data file and store in arrays - this is price data here//
	f_close(source,fin);															//close data file
	ret(p, end, rets);
	lensz = (int)(sharplist[gg][2][ii-tachy] );
	zscore(lensz, p, sumv, varv, end, zscores, stdevv, m_avev);

	//** COPY CUDA VARIABLES FROM CPU (HOST) TO GPU (DEVICE) - USE ASYNC TRANSFER FOR MORE SPEED SO CPU DOES NOT HAVE TO WAIT FOR GPU TO FINISH OPERATION AND CAN PROCEED FURTHER IN THE MAIN PROGRAM**
	hipMemcpyAsync(zscores_d, zscores, (int)(end)*sizeof(float), hipMemcpyHostToDevice,0);
	hipMemcpyAsync(rets_d, rets, (int)(end)*sizeof(float), hipMemcpyHostToDevice,0);

	gap2 = (stopf - starto)/STR;
	lenny=stopf-starto;
	dim3 threads; threads.x = THR;  //use 896 threads as per specific GPU device for higher OCCUPANCY/USE OF CARD - trial-and-error via PROFILING
    dim3 blocks; blocks.x = (lenny/threads.x) + 1;
	//kernelSim<<<threads,blocks>>>(zscores_d,rets_d,pnl_d,pos_d,start,stop,a[z]);

	// allocate and initialize an array of stream handles
    hipStream_t *streams = (hipStream_t *) malloc(STR * sizeof(hipStream_t));

	//** Create Streams for Concurrency or Multi-Streaming - now we will call several KERNELS simultaneously**
	for(int i = 0; i < STR; i++) hipStreamCreate(&(streams[i]));

	//** CALL GPU FUNCTION/KERNEL HERE FOR MODEL PARAMETER SWEEP TO GENERATE OS_SAMPLE RESULTS**THIS IS THREAD REDUCTION DUE TO CONCURRENCY!
    //kernelSim<<<threads,32>>>(zscores_d,rets_d,pnl_d,pos_d,start,stop,(float)(a[z]),lens);
	for (i = 0; i < STR; i++)
		{
			kernelSim<<<32,threads,0,streams[i]>>>(zscores_d,rets_d,pnl_d,pos_d,starto+(i*gap2),starto+((i+1)*gap2),(float)(sharplist[gg][1][ii-tachy]),lensz);
			if (i == STR-1) kernelSim<<<32,threads,0,streams[i]>>>(zscores_d,rets_d,pnl_d,pos_d,starto+(i*gap2),stopf,(float)(sharplist[gg][1][ii-tachy]),lensz);
		}

	//SYNC up STREAMS before copying back data to CPU
	hipStreamSynchronize(streams[STR-1]);

	//** COPY CUDA VARIABLES/RESULTS FROM GPU (DEVICE) BACK TO CPU (HOST) - MUST WAIT FOR GPU OPERATION/FUNCTION TO FINISH HERE SINCE LOW ASYNC/CONCURRENCY ON NON_TESLA GPU DEVICES**
	hipMemcpy(pos, pos_d, (int)(end)*sizeof(float)/*stop-start*/, hipMemcpyDeviceToHost);
	hipMemcpy(pnl, pnl_d, (int)(end)*sizeof(float), hipMemcpyDeviceToHost);

	//** Destroy Streams for Concurrency or Multi-Streaming - now we will RELEASE resources back to GPU**
	for(int i = 0; i < STR; i++) hipStreamDestroy(streams[i]);


	//for(i=starto;i<stopf;i++){														//OUT-OF-SAMPLE runs for old CPU CODE - NOW WE'RE USING GPU INSTEAD FOR MORE SPEED**
	//
	//	if(zscores[i] > sharplist[gg][1][ii-tachy]) pos[i] = 1.00;
	//	if(zscores[i] < -sharplist[gg][1][ii-tachy]) pos[i] = -1.00;
	//	pnl[i] = (pos[i] * rets[i]);}




		sharpep(pnl, sumip, varip, starto, stopf, stdevip, m_aveip, sharpp);
		//ocum[gg][0][ii] = cumpnl[stopf-1];
		//ocum[gg][1][ii] = cumdolls[stopf-1];
		ocum[gg][2][ii] = sharpp[stopf-1];
		if (sharpp[stopf-1] > 0.00) combos = combos + 1;
		sharpies = sharpies + sharpp[stopf-1]/(peri[gg]);
		counters=counters+1;
		mat =(((((ii+0)*(double)(STEP)))+LOOKBACK)/260);
		if (mat >= ((double)(end)/260)) mat = ((double)(end)/260);
		if (stop>=end) mat = ((double)(end)/260);
		printf("\nOS PNL: Market%d-%s Period%d", gg,marks[(gg-1)+(dd*gapo)],ii);
		printf("\nOS-Sharpe\t%.2f", sharpp[stopf-1]);
		printf("\nOS-Param1\t%.2f", sharplist[gg][1][ii-tachy]);
		printf("\nOS-Param2\t%.0f", sharplist[gg][2][ii-tachy]);
		//printf("\nOS-Cum Ret\t%.2f%%", cumpnl[stopf-1]*100);
		//printf("\nOS-Ann Ret\t%.2f%%", (cumpnl[stopf-1]*100)/(LOOKBACK/260));
		//printf("\nOS-Ann Vol\t%.2f%%", ABS(((cumpnl[stopf-1]*100)/(LOOKBACK/260))/sharpp[stopf-1]));
		printf("\nNum of Years: %.2f thru %.2f of %.2f total\n", (((((ii-1)*(double)(STEP)))+LOOKBACK)/260),mat,((double)(end)/260));
		fprintf(recon,"\nOS PNL: Market%d-%s Period%d", gg,marks[(gg-1)+(dd*gapo)],ii);
		fprintf(recon,"\nOS-Sharpe\t%.2f", sharpp[stopf-1]);
		fprintf(recon,"\nOS-Param1\t%.2f", sharplist[gg][1][ii-tachy]);
		fprintf(recon,"\nOS-Param2\t%.0f", sharplist[gg][2][ii-tachy]);
		//fprintf(recon,"\nOS-Cum Ret\t%.2f%%", cumpnl[stopf-1]*100);
		//fprintf(recon,"\nOS-Ann Ret\t%.2f%%", (cumpnl[stopf-1]*100)/(LOOKBACK/260));
		//fprintf(recon,"\nOS-Ann Vol\t%.2f%%", ABS(((cumpnl[stopf-1]*100)/(LOOKBACK/260))/sharpp[stopf-1]));
		fprintf(recon,"\nNum of Years: %.2f thru %.2f of %.2f total\n", (((((ii-1)*(double)(STEP)))+LOOKBACK)/260),mat,((double)(end)/260));

	for(i=starto;i<stopf;i++) {
		fprintf(ferr,"%ld\t %10.6lf\t %10.3lf\t %10.2lf\t %10.5lf\t %10.5lf\t %10.5lf\n",dt[i],p[i],zscores[i],pos[i],rets[i],pnl[i],sharpp[i]);
		fprintf(ferri,"%ld\t %10.6lf\t %10.3lf\t %10.2lf\t %10.5lf\t %10.5lf\t %10.5lf\n",dt[i],p[i],zscores[i],pos[i],rets[i],pnl[i],sharpp[i]+ocum[gg][2][ii-1]); } //


	f_close(dest,ferr);							//close output file

	if (sharpp[stopf-1] > 0.00) winos = winos+ii/(peri[gg]);
	/*ocum[gg][0][ii] = cumpnl[stopf-1];
	ocum[gg][1][ii] = cumdolls[stopf-1];
	ocum[gg][2][ii] = sharpp[stopf-1];*/
	tachy=0;

	free(hi);free(lo);free(p);free(price);free(smooth);free(detrender);free(period);free(qu);free(iu);free(ji);free(jq);free(ib);free(qb);free(sib);free(re);free(im);free(sre);free(sim);free(speriod);free(smperiod);free(qc);free(ic);free(ric);free(intperiod);free(sig);free(nois);free(snr);
	hipHostFree(rets);hipHostFree(zscores);free(cumpnl);free(op);free(sharp);free(sumi);free(vari);free(stdevi);free(m_avei);//hipFree(zscores);hipFree(rets);hipFree(pnl);hipFree(pos);
	free(sumv);free(varv);free(stdevv);free(m_avev);free(dolls);free(cumdolls);free(sharpp);free(sumip);free(varip);free(stdevip);free(m_aveip);hipHostFree(pos);hipHostFree(pnl);
	hipFree(zscores_d);hipFree(pnl_d);hipFree(pos_d);hipFree(rets_d);

	}

	f_close(desta,ferri);						//close output file

}

	//hipDeviceSynchronize();
	ss = dd;
	ss = clock();
	diff = (((float)ss - (float)ff) / 1000000.0F ) * 1;
	printf("\n\n%.3f min..Avg I-MaxSharpe is %.2f..+I-Sharpes are %d of %d combos or %.0f%%..\n+%d Win Markets out of %i..Avg All I-Sharpes is %.2f\n\n",diff/60.0,avv/NUMI,combo,counter,(((double)(combo)/(double)(counter))*100),wins,NUMI,sharpie/(double)(counter));
	fprintf(recon,"\n\n%.3f min..Avg I-MaxSharpe is %.2f..+I-Sharpes are %d of %d combos or %.0f%%..\n+%d Win Markets out of %i..Avg All I-Sharpes is %.2f\n\n",diff/60.0,avv/NUMI,combo,counter,(((double)(combo)/(double)(counter))*100),wins,NUMI,sharpie/(double)(counter));

	printf("\n\n%.3f min..+O-Sharpes are %d of %d combos or %.0f%%..\n+%d Win Markets out of %i..Avg All O-Sharpes is %.2f\n\n",diff/60.0,combos,counters,(((double)(combos)/(double)(counters))*100),winos,NUMI,sharpies/(double)(NUMI));//(double)(counters)); //NUMI
	fprintf(recon,"\n\n%.3f min..+O-Sharpes are %d of %d combos or %.0f%%..\n+%d Win Markets out of %i..Avg All O-Sharpes is %.2f\n\n",diff/60.0,combos,counters,(((double)(combos)/(double)(counters))*100),winos,NUMI,sharpies/(double)(NUMI)); //NUMI

	f_close(destr,recon); 
	//hipDeviceReset();
	hipSetDevice(dd);
       hipDeviceReset();
	diff=0.00f;



}

	//**RESET GPU DEVICE**//	
	for (i = 0; i < GPUn; i++)
    	{
        hipSetDevice(i);
        hipDeviceReset();
    	}


}



long f_line(FILE *f)  {               /*count the lines*/
	char ch;
	long count;
	count=1;
	while( (ch = getc(f)) != EOF)
	  	if(ch == '\n')
			count++;
    fseek(f,0L,SEEK_SET);
	return(count);	}

FILE * f_openr( char *filer)   {
	FILE *f;
	f=fopen(filer,"r");
	if(f == NULL)	{
		perror("Error");
		fprintf(stderr," file %s cannot be open under r mode \n",filer);
		exit(1);	}
	return f;	}


FILE * f_openw( char *filer)      {
	FILE *f;
	f=fopen(filer,"w");
	if(f == NULL)	{
		perror("Error");
		fprintf(stderr," file %s cannot be open under w mode \n",filer);
		exit(1);	}
	return f;	}



void f_close(char *filer,FILE *f){
	if(fclose(f) !=0)	{
		printf("error in closing file %s \n",filer);
		exit(2);	}	}



void snf(double *op, double *lo, double *hi, long end, double *price, double *smooth, double *detrender, double *period, double *qu, double *iu, double *ji, double *jq, double *ib, double *qb, double *sib, double *sqb, double *re, double *im, double *sre, double *sim, double *speriod, double *smperiod, double *qc, double *ic, double *ric, int *intperiod, double *sig, double *nois, double *snr){

	int i,count;

	for(i=1;i<end;i++){
		if(i>5) price[i] = ((op[i]+op[i])/2);
		if(i>5) smooth[i] = (4*price[i] + 3*price[i-1] + 2*price[i-2] + price[i-3]) / 10;
		if(i>5) detrender[i] = (.0962*smooth[i] + .5769*smooth[i-2] - .5769*smooth[i-4] - .0962*smooth[i-6])*(.075*period[i-1] + .54);
		if(i>5) iu[i] = detrender[i-3];
		if(i>5) qu[i] = (.0962*detrender[i] + .5769*detrender[i-2] - .5769*detrender[i-4] - .0962*detrender[i-6])*(.075*period[i-1] + .54);
		if(i>5) ji[i] = (.0962*iu[i] + .5769*iu[i-2] - .5769*iu[i-4] - .0962*iu[i-6])*(.075*period[i-1] + .54);
		if(i>5) jq[i] = (.0962*qu[i] + .5769*qu[i-2] - .5769*qu[i-4] - .0962*qu[i-6])*(.075*period[i-1] + .54);
		if(i>5) ib[i] = iu[i] - jq[i];
		if(i>5) qb[i] = qu[i] + ji[i];
		if(i>5) sib[i] = .2*ib[i] + .8*ib[i-1];
		if(i>5) sqb[i] = .2*qb[i] + .8*qb[i-1];
		if(i>5) re[i] = sib[i]*sib[i-1]+sqb[i]*sqb[i-1];
		if(i>5) im[i] = sib[i]*sqb[i-1]-sqb[i]*sib[i-1];
		if(i>5) sre[i] = .2*re[i] + .8*re[i-1];
		if(i>5) sim[i] = .2*im[i] + .8*im[i-1];
		if(sim[i]!=0&&sre[i]!=0&&i>5) period[i] = 360/atan(sim[i]/sre[i]);
		if(i>5&&period[i]>period[i-1]*1.5) period[i]=1.5*period[i-1];
		if(i>5&&period[i]<period[i-1]*.67) period[i]=.67*period[i-1];
		if(i>5&&period[i]<6) period[i]=6;
		if(i>5&&period[i]>50) period[i]=50;
		if(i>5) speriod[i] = .2*period[i] + .8*period[i-1];
		if(i>5) smperiod[i] = .33*speriod[i] + .67*smperiod[i-1];
		if(i>5) intperiod[i] = (int)(smperiod[i]*.5);
		if(i>5) qc[i] = .5*(smooth[i]-smooth[i-2])*(.1759*smperiod[i]+.4607);
			ic[0]=0;
		for(count=0;count<=intperiod[i]-1;count++){
			ic[i] = qc[count]+ic[i];}
		if(i>5&&(intperiod[i])!=0) ic[i]=1.57*ic[i]/(intperiod[i]);
		if(i>5) sig[i] = ic[i]*ic[i] + qc[i]*qc[i];
		if(i>5) nois[i] = .1*(hi[i]-lo[i])*(hi[i]-lo[i])*.25+.9*nois[i-1];
		if(i>5&&nois[i]!=0&&sig[i]!=0) snr[i] = .33*(10*log(sig[i]/nois[i])/log(10.00))+.67*snr[i-1];  // REF LEVEL is 8 by 13 //
		//snr[i] = snr[i]*.5;

	}
	return;
}


//Zscore FUNC//
void zscore(int lens, double *op, double *sumv, double *varv, long end, float *zscores, double *stdevv, double *m_avev){


	int i,h;
	varv[0] = 0;
	sumv[0] = 0;
	m_avev[0] = 0;
	stdevv[0] = 0;
	for(i=1;i<end;i++) sumv[i]=sumv[i-1]+op[i];
	for(i=1;i<end;i++){
		if(i>=lens) m_avev[i]=(sumv[i]-sumv[i-lens])/lens;
		if(i>=lens) for(h=0;h<lens;h++) varv[i]+=(((op[i-h]-m_avev[i])*(op[i-h]-m_avev[i]))/(lens-1));
		if(i>=lens) stdevv[i]=sqrt(varv[i]);	//using square root function from math library here//
	    if(i>=lens) zscores[i]=(op[i] - m_avev[i]) / stdevv[i];	}

	return;
}


//Get RETURNS function from market closing price//
void ret(double *p, long end, float *rets){


	int i=0;
	rets[0] = 0;
	//if(i>0) {
		for(i=1;i<end;i++) rets[i]=(p[i] - p[i-1]) / p[i-1];
	//else {rets[i] = 0.00; }

	return;
}

//Calc SHARPE RATIO FUNC of PNL returns//
void sharpe(float *pnl, double *sumi, double *vari, long end, double *stdevi, double *m_avei, double *sharp){


	int i,h;
	vari[0] = 0;
	sumi[0] = 0;
	m_avei[0] = 0;
	stdevi[0] = 0;
	for(i=1;i<end;i++) sumi[i]=sumi[i-1]+pnl[i];
	for(i=1;i<end;i++){
		if(i>1) m_avei[i]=((sumi[i])/i);
		if(i>1) for(h=0;h<i;h++) vari[i]+=(((pnl[i-h]-m_avei[i])*(pnl[i-h]-m_avei[i]))/(i-1));
		if(i>1) stdevi[i]=sqrt(vari[i]);	//using square root function from math library here//
	    if(i>1) sharp[i]=((m_avei[i]*260) / (stdevi[i]*sqrt(260.00)));	}

	return;
}

void sharpep(float *pnl, double *sumip, double *varip, int start, int stop, double *stdevip, double *m_aveip, double *sharpp){


	int i,h;
	varip[0] = 0;
	sumip[0] = 0;
	m_aveip[0] = 0;
	stdevip[0] = 0;
	for(i=start;i<stop;i++) sumip[i]=sumip[i-1]+pnl[i];
	for(i=start;i<stop;i++){
		if(i>start) m_aveip[i]=((sumip[i])/i);
		if(i>start) for(h=0;h<i;h++) varip[i]+=(((pnl[i-h]-m_aveip[i])*(pnl[i-h]-m_aveip[i]))/(i-1));
		if(i>start) stdevip[i]=sqrt((varip[i]));	//using square root function from math library here//
	    if(i>start) sharpp[i]=((m_aveip[i]*260) / (stdevip[i]*sqrt(260.00)));	}

	return;
}


__global__ void kernelSim(float *zscores_d,float *rets_d,float *pnl_d,float *pos_d,int start,int stop,float zcut,int lens){



	//float zcut = *a_d;
	//int starty = *start_d;
	//int stoppy = *stop_d;
	//int opt = 0;
	const float buy =1.00f;
	const float sell=-1.00f;
	const float flat=0.00f;
	//const float trans=0.0002f;

	//const float scut = zcut;
	//const float scut = floorf(zcut * 1000) / 1000;   /* Result: 37.77 */



	//Thread index
    unsigned int      tid = blockDim.x * blockIdx.x + threadIdx.x;
    //Total number of threads in execution grid
    //const int THREAD_N = blockDim.x * gridDim.x;
	unsigned int THREAD_N = blockDim.x * gridDim.x;

	// __syncthreads();

    //No matter how small is execution grid or how large OptN is,
    //exactly OptN indices will be processed with perfect memory coalescing


	for(int opt = tid+start; opt < stop; opt += THREAD_N){
	//if(tid < stoppy){
		if(zscores_d[opt] > zcut && opt >=lens) pos_d[opt] = buy;
		if(zscores_d[opt] < -zcut && opt >=lens) pos_d[opt] = sell;
		if(opt >=lens && (pos_d[opt]==buy || pos_d[opt]==sell)) pnl_d[opt] = __fmul_rn(pos_d[opt],rets_d[opt]);
		else {pnl_d[opt] = flat; pos_d[opt] = flat; }

	}


}
