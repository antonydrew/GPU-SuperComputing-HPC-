#include <stdio.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

//CUDA Kernel Device code - Computes the vector addition of 10 to each iteration i
__global__ void kernelTest(int* i, int length){

    unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;

    if(tid < length)
        i[tid] = i[tid] + 10; }

 /* This is the main routine which declares and initializes the integer vector, moves it to the device, launches kernel
 * brings the result vector back to host and dumps it on the console. */
int main(void){

	// Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

	int cumsum[200]={0},x=0;
    int length  = 100;
	printf("[Vector multiplication of %d elements]\n", length);

	// Allocate the host input vector A
    int* i = (int*)malloc(length*sizeof(int));

    for(int x=0;x<length;x++)
        i[x] = x;

	 // Allocate the device input vector 
    int* i_d;
    err=hipMalloc((void**)&i_d,length*sizeof(int));

	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device matrix  (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); }

	// Copy the host input vectors A and B in host memory to the device input vectors in
	printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(i_d, i, length*sizeof(int), hipMemcpyHostToDevice);

	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy matrix from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); }

	// Launch the Vector Add CUDA Kernel
    dim3 threads; threads.x = 256;
    dim3 blocks; blocks.x = (length/threads.x) + 1;
    kernelTest<<<threads,blocks>>>(i_d,length);

	err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorMultiply kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); }

	 // Copy the device result vector in device memory to the host result vector
	printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(i, i_d, length*sizeof(int), hipMemcpyDeviceToHost);

	 if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy matrix from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); }

    for(int x=0;x<length;x++)
        printf("%d\t",i[x]);

	// Verify that the result vector is correct
    for (int x = 1; x <= length; ++x)
    {     
		cumsum[x] = cumsum[x-1]+i[x]; }

	if (cumsum[length-1]+i[0] != 5950)
        {
            fprintf(stderr,"Result verification failed at element %i!\n", cumsum[length-1]);
            exit(EXIT_FAILURE); }
	
	// Free host and device memory
    free(i); hipFree(i_d);
  
    // Reset the device and exit
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE); }

    printf("Done\n");
	return; }