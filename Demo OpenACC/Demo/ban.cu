#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <io.h>
#include <stdlib.h>
#include <string.h> 
#include <float.h>
#include "utility.h"
#include "utility.c"
//#include "Trade.cuh"
#include <time.h>
#include <malloc.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
//#include <unistd.h>
#include <omp.h>
//#include "hip/hip_runtime_api.h"
//#include "helper_functions.h"

#pragma warning( disable : 4996 )

int main () {

  // banner
  printf ("\n\n     Coding Exercise 3\n");
  printf (    "     ===============================\n");
  printf (  "\n     Matrix-Matrix Multiplication\n");
  printf (    "     PGI / OpenACC acceleration \n");

  // define parameters 
  int n = 1024;  // matrix dimension
  
  // allocate arrays
  double *a = (double *) malloc ( n*n*sizeof(double) );
  double *b = (double *) malloc ( n*n*sizeof(double) );
  double *c = (double *) malloc ( n*n*sizeof(double) );
  
  // initialize data
  for ( int row = 0; row<n; row++ ) {
    for ( int col = 0; col<n; col++ ) {
      // data is in row-major format
      a[row*n+col] = sin( 0.01*col ) + cos( 0.013*row );
      b[row*n+col] = sin( 0.017*col ) + cos( 0.03*row );
    }
  }

//#pragma acc data copy( c[0:n*n] )
#pragma acc kernels loop copy( c[0:n*n] )
  for ( int i = 0; i<n*n; i++ ) {
      c[i] = 0.0;
  }

  // record start time - use cuda events, accurate
  double t_start = omp_get_wtime();

#pragma acc data copyin(a[0:n*n],b[0:n*n]) copyout(c[0:n*n])
{
  // PERFORM MULTIPLICATION

  // loop over output rows
  #pragma acc kernels
  {

  #pragma acc loop independent
  for ( int row=0; row<n; row++ ) {

    // loop over output columns
    #pragma acc loop independent
    for ( int col=0; col<n; col++ ) {

      // initialize output result to zero
      double val = 0;

      // loop over inner dimension
      #pragma acc loop independent
      for ( int k=0; k<n; k++ ) {
        // sum
	val += a[row*n+k] * b[k*n+col];
	
      }
      c[row*n+col] = val;
	  //printf("%4.4f\n", val);
    }
  }
  }
}

  // compute elapsed time
  double et = omp_get_wtime() - t_start;

  // report results
  printf("\n     reference (768,768) = %4.4f \n", c[768*n+768]);
  printf(  "     elapsedTime         = %4.4f seconds\n", et);  // hipEventElapsedTime is in milliseconds
  printf(  "     gigaflops achieved  = %4.4f Gflops/s\n\n\n", 2.0e-9*n*n*n/et); // 2( * and + ) *n (inner dimension)*n^2(result size)/(time in s.)

  system("pause");

}