#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <io.h>
#include <stdlib.h>
#include <string.h> 
#include <float.h>
#include "utility.h"
#include "utility.c"
//#include "Trade.cuh"
#include <time.h>
#include <malloc.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
//#include "hip/hip_runtime_api.h"
//#include "helper_functions.h"

#pragma warning( disable : 4996 )

//Declaring macros and constants in pre-processor - STEP A//INSERT NEW STUFF DOWN HERE EACH TIME START SUB-FUNCTION******PRE-PROCESSOR AREA*******//
#define ABS(X) (X>=0?X:-X)
#define MAX(X,Y) (X>=Y?X:Y)
#define MIN(X,Y) (X<=Y?X:Y)
#define SIGN(X) (X>=0?(X==0?0:1):-1)
#define ROUND(X,Y) ((X>=0?(X<<1)+Y:(X<<1)-Y)/(Y<<1))*Y 

//Change path below for UNIX "c://usr"
#define PATH "F:\\"			
#define LOOKBACK 1597		// 1597-987-610-377-144-89 fibos rolling optimization historical period
#define STEP 377			// or 89 fibos step forward in time period for next rolling optimization
#define NUMI 27				//up to 27 number of markets 
   


//INT MAIN//INSERT NEW STUFF HERE EACH TIME START SUB-FUNCTION*******MAIN AREA****//Declare each new variable here - initializing and declaring space/memory for return arrays of variables or output we want****STEP B//
int main(int argc, char **argv){
//void main (int argc, char *argv[]){  
	
	FILE *recon, *fin, *ferr, *fins,*ferri;
	int *intperiod,*start_h,*stop_h,c, lens=0, combos=0,counters=0,starto=0,tachy=0,lenny=0;
	long *dt; 
	float *zscores_d, *pnl_d, *pos_d, *rets_d,*a_d,*a_h,*pos,*pnl,*zscores,*rets; 
	double *op,*hi,*lo,*p,*price, *smooth, *detrender, *period, *qu, *iu, *ji, *jq, *ib, *qb, *sib, *sqb,*re, *im,*sre,*sim,*speriod,*smperiod,*qc,*ic,*ric,*sig,*nois,*snr, *cumpnl, *sharp;
	double *sumi, *vari, *stdevi, *m_avei,*sumv, *varv, *stdevv, *m_avev, *dolls, *cumdolls, *sumip, *varip, *stdevip, *m_aveip, *sharpp;
	int i=0,combo=0,ii=0,zz=0,wins=0,counter=0,start=1,stop=0,*start_d,*stop_d,startf=0,stopf=0,beg=1,high=0,m=0,mm=0,gg=0; char desty[50],dest[50],desta[50],tmp[50],strs[50],foldr[50],fnum[50],fnums[50],dir[50]; int peri[100] = { { 0 } };
	
	double pp[] = { 42000.00, 42000.00, 50.00, 20.00, 100.00, 100.00, 10.00, 25.00, 5.00, 1000.00, 1000.00, 2000.00, 1000.00, 2500.00, 100.00, 25000.00, 5000.00, 50.00, 100000.00, 125000.00, 125000.00, 125000.00, 62500.00, 50.00, 1000.00, 10000.00, 50.00 };
	char *marks[] = {"RBOB","HO","SP", "ND", "EMD", "TF", "FESX", "FDAX", "NK", "US", "TY", "TU", "FGBL", "ED", "GC", "HG", "SI", "PL", "AD", "EC", "SF", "JY", "BP", "S", "CL", "NG", "C"};
	char *dfiles[] = {PATH"data0.dat",PATH"data1.dat",PATH"data2.dat",PATH"data3.dat",PATH"data4.dat",PATH"data5.dat","c:\\data6.dat",PATH"data7.dat",PATH"data8.dat",PATH"data9.dat",PATH"data10.dat",PATH"data11.dat",PATH"data12.dat",PATH"data13.dat",
		PATH"data14.dat",PATH"data15.dat",PATH"data16.dat",PATH"data17.dat",PATH"\\data18.dat",PATH"data19.dat",PATH"data20.dat",PATH"data21.dat",PATH"data22.dat",PATH"data23.dat",PATH"data24.dat",PATH"data25.dat",PATH"data26.dat"}; 
	
	clock_t t1, t2; float diff; char sources[60],source[60],line[100]; long end, endf;
	double a[] = { 1.25, 1.50 };		//array holder for parameter combinations later on aka "parameter sweeps" which GPU can greatly speed up// a[] is # standard deviations//
	double b[] = { 21.00, 34.00 };	
	//double b[] = { 3.0, 5.0, 8.0, 10.0 };			
	double lensa=sizeof(a)/sizeof(double); double maxi=0.00,mat=0.00;
	double lensb=sizeof(b)/sizeof(double);
	double lensc=lensa * lensb;				//number of parameter combinations
	double sharplist[100][7][100] = { { 0 } };
	double table[100][5]= { { 0 } }; double ocum[100][3][100]= { { 0 } };
	int z=0, j=0, winos=0,lensz=0; double sumss[100] = { { 0 } }; double avv=0.00, sharpie=0.00, sharpies=0.00;
	t1 = clock();
	recon=f_openw(PATH"recon.dat");	
		
for (gg = 1; gg <= NUMI; gg++) {		// top loop for number of market data files passed thru dfiles[] // must change NUMI in #def as add number of markets // should I use STRUCT instead to store file names?
	
	sprintf(sources,dfiles[gg-1]);			//find and open price data files to get lengths for periodicities//
	fins=f_openr(sources);  
	endf=f_line(fins);
	endf--; 
	peri[gg] = (int)(((endf-LOOKBACK)/STEP)+1); //number of rolling periods in each data set for rolling optimization (aka moving average)//
	f_close(sources,fins); 
	sprintf(fnums, "%d", gg);
	strcpy(desta, PATH"OSrunALL");		//output directory for out-of-sample tests for all combined tests per market
	strcpy(foldr, ".dat"); 
	strcat(desta,"-");  
	strcat(desta,fnums);  
	strcat(desta,"-");  
	strcat(desta,marks[gg-1]);  
	strcat(desta,foldr); 
	ferri=f_openw(desta);	

 for (ii = 1; ii <= peri[gg] ; ii++) {	// loop is for periodicity - so 30yrs of price data divided into sub-units for rolling optimization (aka parameter sweeps)
  for (z = 0; z < lensa; z++) {			// 2 nested for loops for parameter sweep or combination of arrays a[] and b[]//
   for (j = 0; j < lensb; j++) {	
	
	lens = (int)(b[j]);
	mm=(lensa*z)+j;
	sprintf(strs, "%d", mm);		
	sprintf(fnum, "%d", gg);
	sprintf(tmp, "%d", ii);		
	strcpy(desty, PATH"ISout");		//output directory for in-sample tests
	strcpy(foldr, ".dat"); 
	strcat(desty,strs);  
	strcat(desty,"-");  
	strcat(desty,fnum);  
	strcat(desty,"-");  
	strcat(desty,tmp); 
	strcat(desty,"-");  
	strcat(desty,marks[gg-1]);  
	strcat(desty,foldr); 
	ferr=f_openw(desty);				//find and open output file//	
	sprintf(source,dfiles[gg-1]);			//find and open price data file//
	fin=f_openr(source);  
	end=f_line(fin);
	end--; 
	start = beg + (STEP*(ii-1));		//start stop dates for inner loop ii for rolling period optimizations
	stop = LOOKBACK + (STEP*(ii-1));  

	dt=(long*) calloc(end+1,sizeof(long));
	op=(double*) calloc(end+1,sizeof(double));
	hi=(double*) calloc(end+1,sizeof(double));
	lo=(double*) calloc(end+1,sizeof(double));
	p=(double*) calloc(end+1,sizeof(double));
	price=(double*) calloc(end+1,sizeof(double));
	smooth=(double*) calloc(end+1,sizeof(double));
	detrender=(double*) calloc(end+1,sizeof(double)); 
	period=(double*) calloc(end+1,sizeof(double));
	qu=(double*) calloc(end+1,sizeof(double));
	iu=(double*) calloc(end+1,sizeof(double));
	ji=(double*) calloc(end+1,sizeof(double));
	jq=(double*) calloc(end+1,sizeof(double));
	ib=(double*) calloc(end+1,sizeof(double));
	qb=(double*) calloc(end+1,sizeof(double));
	sib=(double*) calloc(end+1,sizeof(double));
	sqb=(double*) calloc(end+1,sizeof(double));
	re=(double*) calloc(end+1,sizeof(double));
	im=(double*) calloc(end+1,sizeof(double));
	sre=(double*) calloc(end+1,sizeof(double));
	sim=(double*) calloc(end+1,sizeof(double));
	speriod=(double*) calloc(end+1,sizeof(double));
	smperiod=(double*) calloc(end+1,sizeof(double));
	qc=(double*) calloc(end+1,sizeof(double));
	ic=(double*) calloc(end+1,sizeof(double));
	ric=(double*) calloc(end+1,sizeof(double));
	intperiod=(int*) calloc(end+1,sizeof(double));
	sig=(double*) calloc(end+1,sizeof(double));
	nois=(double*) calloc(end+1,sizeof(double));
	snr=(double*) calloc(end+1,sizeof(double));
	

	//** ALLOCATE SPACE FOR MEMORY FOR CUDA-RELATED HOST VARIABLES - USE PINNED/SHARED MEMORY FOR MORE SPEED!!! **

	//rets=(float*) calloc(end+1,sizeof(float));
	hipHostAlloc(&rets, (int)(end)*sizeof(float), hipHostMallocDefault);
	//memset(rets, 0, (int)(end)*sizeof(float));
	dolls=(double*) calloc(end+1,sizeof(double));
	cumdolls=(double*) calloc(end+1,sizeof(double));
	//pos=(float*) calloc(end+1,sizeof(float));
	hipHostAlloc(&pos, (int)(end)*sizeof(float), hipHostMallocDefault);
	//memset(pos, 0, (int)(end)*sizeof(float));
	//pnl=(float*) calloc(end+1,sizeof(float));
	hipHostAlloc(&pnl, (int)(end)*sizeof(float), hipHostMallocDefault);
	//memset(pnl, 0, (int)(end)*sizeof(float));
	cumpnl=(double*) calloc(end+1,sizeof(double));
	//zscores=(float*) calloc(end+1,sizeof(float));
	hipHostAlloc(&zscores, (int)(end)*sizeof(float), hipHostMallocDefault);
	//memset(zscores, 0, (int)(end)*sizeof(float));

	sharpp=(double*) calloc(end+1,sizeof(double));
	sumip=(double*) calloc(end+1,sizeof(double));
	varip=(double*) calloc(end+1,sizeof(double));
	stdevip=(double*) calloc(end+1,sizeof(double));
	m_aveip=(double*) calloc(end+1,sizeof(double));
	sharp=(double*) calloc(end+1,sizeof(double));
	sumi=(double*) calloc(end+1,sizeof(double));
	vari=(double*) calloc(end+1,sizeof(double));
	stdevi=(double*) calloc(end+1,sizeof(double));
	m_avei=(double*) calloc(end+1,sizeof(double));
	sumv=(double*) calloc(end+1,sizeof(double));
	varv=(double*) calloc(end+1,sizeof(double));
	stdevv=(double*) calloc(end+1,sizeof(double));
	m_avev=(double*) calloc(end+1,sizeof(double));
	
	
	//** ALLOCATE SPACE FOR MEMORY FOR CUDA-RELATED DEVICE VARIABLES**

	hipMalloc((void**)&zscores_d, (int)(end)*sizeof(float));
	//hipMemset(zscores_d, 0, (int)(end)*sizeof(float));
	hipMalloc((void**)&pos_d, (int)(end)*sizeof(float));
	hipMemset(pos_d, 0, (int)(end)*sizeof(float));
	hipMalloc((void**)&pnl_d, (int)(end)*sizeof(float));
	hipMemset(pnl_d, 0, (int)(end)*sizeof(float));
	hipMalloc((void**)&rets_d, (int)(end)*sizeof(float));
	//hipMemset(rets_d, 0, (int)(end)*sizeof(float));
	/*hipMalloc((void**)&a_d, 1);
	hipMalloc((void**)&start_d, 1);
	hipMalloc((void**)&stop_d, 1);*/

	
	
//INSERT NEW STUFF HERE EACH TIME START SUB-FUNCTION*************///CALLING function//STEP C//
	
	i=0;
	while(fgets(line,100,fin)>0){
		sscanf(line,"%ld %lf %lf %lf %lf",&dt[i],&op[i],&hi[i],&lo[i],&p[i]);i++;}  //scan lines from data file and store in arrays - this is price data here//
	f_close(source,fin);															//close data file

	//Using or CALLING function here//  DO NOT NEED TO DEFINE INPUTS-OUTPUTS here - that is done at bottom down BELOW!!//

	ret(p, end, rets);
	//snf(op, lo, hi, end, price, smooth, detrender, period, qu, iu, ji, jq, ib, qb, sib, sqb, re, im, sre, sim, speriod, smperiod, qc, ic, ric, intperiod, sig, nois, snr);
	zscore(lens, p, sumv, varv, end, zscores, stdevv, m_avev);
	m = (lensa*z)+j;
	
	//** COPY CUDA VARIABLES FROM CPU (HOST) TO GPU (DEVICE) - USE ASYNC TRANSFER FOR MORE SPEED SO CPU DOES NOT HAVE TO WAIT FOR GPU TO FINISH OPERATION AND CAN PROCEED FURTHER IN THE MAIN PROGRAM**
	hipMemcpyAsync(zscores_d, zscores, (int)(end)*sizeof(float), hipMemcpyHostToDevice,0);
	hipMemcpyAsync(rets_d, rets, (int)(end)*sizeof(float), hipMemcpyHostToDevice,0);


	lenny=stop-start;
	dim3 threads; threads.x = 896;		//use 896 threads as per specific GPU device for higher OCCUPANCY/USE OF CARD - trial-and-error via PROFILING
    //dim3 blocks; blocks.x = ((int)(end)/threads.x) + 1;
	//kernelSim<<<threads,blocks>>>(zscores_d,rets_d,pnl_d,pos_d,start,stop,a[z]);

	//** CALL GPU FUNCTION/KERNEL HERE FOR MODEL PARAMETER SWEEP TO GENERATE IN_SAMPLE RESULTS**
    kernelSim<<<threads,112>>>(zscores_d,rets_d,pnl_d,pos_d,start,stop,(float)(a[z]),lens);

	//** COPY CUDA VARIABLES/RESULTS FROM GPU (DEVICE) BACK TO CPU (HOST) - MUST WAIT FOR GPU OPERATION/FUNCTION TO FINISH HERE SINCE LOW ASYNC/CONCURRENCY ON NON_TESLA GPU DEVICES**
	hipMemcpy(pos, pos_d, (int)(end)*sizeof(float)/*stop-start*/, hipMemcpyDeviceToHost);
	hipMemcpy(pnl, pnl_d, (int)(end)*sizeof(float), hipMemcpyDeviceToHost);
	
	

	//for(i=start;i<stop;i++){														//IN-sample rolling optimization for old CPU CODE - NOW WE'RE USING GPU INSTEAD FOR MORE SPEED**
	//	
	//	if(zscores[i] > a[z]) pos[i] = 1.00;														
	//	if(zscores[i] < -a[z]) pos[i] = -1.00;			
	//	pnl[i] = (pos[i] * rets[i]); }

		
		sharpep(pnl, sumip, varip, start, stop, stdevip, m_aveip, sharpp);
		table[m][0] = m;
		table[m][1] = a[z];
		table[m][2] = b[j];
		table[m][3] = sharpp[stop-1];
		sharpie = sharpie + sharpp[stop-1];//end?
		if (table[m][3] > 0.00) combo = combo + 1; 
		counter=counter+1;
		//table[m][4] = cumpnl[stop-1];
		printf("\nIS Test%.0f Market%d-%s Period%d", table[m][0],gg,marks[gg-1],ii);		
		printf("\nSharpe\t%.2f", table[m][3]);
		printf("\nParam1\t%.2f", table[m][1]);
		printf("\nParam2\t%.0f", table[m][2]);
		//printf("\nCum Ret\t%.2f%%", table[m][4]*100);
		//printf("\nAnn Ret\t%.2f%%", (table[m][4]*100)/(LOOKBACK/260));
		//printf("\nAnn Vol\t%.2f%%", ABS(((table[m][4]*100)/(LOOKBACK/260))/table[m][3]));
		printf("\nNum of Years: %.2f thru %.2f of %.2f total\n", ((((ii-1)*(double)(STEP)))/260),(((double)(LOOKBACK) + (ii*(double)(STEP)))/260)-(double)(STEP)/260,((double)(end)/260));
		fprintf(recon,"\nIS Test%.0f Market%d-%s Period%d", table[m][0],gg,marks[gg-1],ii);		
		fprintf(recon,"\nSharpe\t%.2f", table[m][3]);
		fprintf(recon,"\nParam1\t%.2f", table[m][1]);
		fprintf(recon,"\nParam2\t%.0f", table[m][2]);
		//fprintf(recon,"\nCum Ret\t%.2f%%", table[m][4]*100);
		//fprintf(recon,"\nAnn Ret\t%.2f%%", (table[m][4]*100)/(LOOKBACK/260));
		//fprintf(recon,"\nAnn Vol\t%.2f%%", ABS(((table[m][4]*100)/(LOOKBACK/260))/table[m][3]));
		fprintf(recon,"\nNum of Years: %.2f thru %.2f of %.2f total\n", ((((ii-1)*(double)(STEP)))/260),(((double)(LOOKBACK) + (ii*(double)(STEP)))/260)-(double)(STEP)/260,((double)(end)/260));
		
	for(i=start;i<stop;i++) {
		fprintf(ferr,"%ld\t %10.6lf\t %10.3lf\t %10.2lf\t %10.5lf\t %10.5lf\t %10.5lf\n",dt[i],p[i],zscores[i],pos[i],rets[i],pnl[i],sharpp[i]); } //	

	f_close(desty,ferr);														//close output file
	
	for (i = 0; i < lensc; i++)													//find best sharpe ratio from table
            {
                if (table[i][3] > maxi) maxi = table[i][3];
                if (maxi == table[i][3]) high=i;				
            }
				
			sharplist[gg][0][ii] = high;										//row of max sharpe recap
			sharplist[gg][6][ii] = table[high][3];								//max sharpe
			sharplist[gg][1][ii] = table[high][1];								//param 1 recap
			sharplist[gg][2][ii] = table[high][2];								//param 2 recap
			sharplist[gg][3][ii] = table[high][4];								//cum ret recap
			sharplist[gg][4][ii] = table[high][0];								//test number recap
			sharplist[gg][5][ii] = gg;											//market number recap		
			
            maxi=0.00;

//ADD IN EACH POINTER VARIABLE HERE - FREEING UP SPACE IN MEMORY*******STEP D//

	//hipDeviceReset();

	free(hi);free(lo);free(p);free(price);free(smooth);free(detrender);free(period);free(qu);free(iu);free(ji);free(jq);free(ib);free(qb);free(sib);free(re);free(im);free(sre);free(sim);free(speriod);free(smperiod);free(qc);free(ic);free(ric);free(intperiod);free(sig);free(nois);free(snr);
	hipHostFree(rets);hipHostFree(zscores);free(cumpnl);free(op);free(sharp);free(sumi);free(vari);free(stdevi);free(m_avei);
	free(sumv);free(varv);free(stdevv);free(m_avev);free(dolls);free(cumdolls);free(sharpp);free(sumip);free(varip);free(stdevip);free(m_aveip);//free(a_h);free(start_h);free(stop_h);
	hipFree(zscores_d);hipFree(pnl_d);hipFree(pos_d);hipFree(rets_d);hipHostFree(pos);hipHostFree(pnl);
	//hipHostFree(zscores_d);hipHostFree(pnl_d);hipHostFree(pos_d);hipHostFree(rets_d);
	
		}
	  }
	
	avv = avv+sharplist[gg][6][ii]/peri[gg];									//avg max sharpe	
	if (sharplist[gg][6][ii] > 0.00) wins = wins+ii/peri[gg];					//winning markets
	printf("\n%.2f Max Sharpe of Market%d-%s Period%d is Test %.0f with STD %.2f and SNR %.0f\n", sharplist[gg][6][ii],gg,marks[gg-1],ii,sharplist[gg][0][ii],sharplist[gg][1][ii],sharplist[gg][2][ii]);	
	fprintf(recon,"\n%.2f Max Sharpe of Market%d-%s Period%d is Test %.0f with STD %.2f and SNR %.0f\n", sharplist[gg][6][ii],gg,marks[gg-1],ii,sharplist[gg][0][ii],sharplist[gg][1][ii],sharplist[gg][2][ii]);	

	
	sprintf(fnum, "%d", gg);
	sprintf(tmp, "%d", ii);		
	strcpy(dest, PATH"OSrun");		//output directory for out-of-sample tests
	strcpy(foldr, ".dat"); 
	strcat(dest,"-");  
	strcat(dest,fnum);  
	strcat(dest,"-");  
	strcat(dest,tmp); 
	strcat(dest,"-");  
	strcat(dest,marks[gg-1]); 
	strcat(dest,foldr); 
	ferr=f_openw(dest);				//find and open output file//	
	sprintf(source,dfiles[gg-1]);	//find and open price data file//
	fin=f_openr(source);  
	end=f_line(fin);
	end--; 
	

	dt=(long*) calloc(end+1,sizeof(long));
	op=(double*) calloc(end+1,sizeof(double));
	hi=(double*) calloc(end+1,sizeof(double));
	lo=(double*) calloc(end+1,sizeof(double));
	p=(double*) calloc(end+1,sizeof(double));
	price=(double*) calloc(end+1,sizeof(double));
	smooth=(double*) calloc(end+1,sizeof(double));
	detrender=(double*) calloc(end+1,sizeof(double)); 
	period=(double*) calloc(end+1,sizeof(double));
	qu=(double*) calloc(end+1,sizeof(double));
	iu=(double*) calloc(end+1,sizeof(double));
	ji=(double*) calloc(end+1,sizeof(double));
	jq=(double*) calloc(end+1,sizeof(double));
	ib=(double*) calloc(end+1,sizeof(double));
	qb=(double*) calloc(end+1,sizeof(double));
	sib=(double*) calloc(end+1,sizeof(double));
	sqb=(double*) calloc(end+1,sizeof(double));
	re=(double*) calloc(end+1,sizeof(double));
	im=(double*) calloc(end+1,sizeof(double));
	sre=(double*) calloc(end+1,sizeof(double));
	sim=(double*) calloc(end+1,sizeof(double));
	speriod=(double*) calloc(end+1,sizeof(double));
	smperiod=(double*) calloc(end+1,sizeof(double));
	qc=(double*) calloc(end+1,sizeof(double));
	ic=(double*) calloc(end+1,sizeof(double));
	ric=(double*) calloc(end+1,sizeof(double));
	intperiod=(int*) calloc(end+1,sizeof(double));
	sig=(double*) calloc(end+1,sizeof(double));
	nois=(double*) calloc(end+1,sizeof(double));
	snr=(double*) calloc(end+1,sizeof(double));
	
	//** ALLOCATE SPACE FOR MEMORY FOR CUDA-RELATED HOST VARIABLES - USE PINNED/SHARED MEMORY FOR MORE SPEED!!! **

	//rets=(float*) calloc(end+1,sizeof(float));
	hipHostAlloc(&rets, (int)(end)*sizeof(float), hipHostMallocDefault);
	//memset(rets, 0, (int)(end)*sizeof(float));
	dolls=(double*) calloc(end+1,sizeof(double));
	cumdolls=(double*) calloc(end+1,sizeof(double));
	//pos=(float*) calloc(end+1,sizeof(float));
	hipHostAlloc(&pos, (int)(end)*sizeof(float), hipHostMallocDefault);
	//memset(pos, 0, (int)(end)*sizeof(float));
	//pnl=(float*) calloc(end+1,sizeof(float));
	hipHostAlloc(&pnl, (int)(end)*sizeof(float), hipHostMallocDefault);
	//memset(pnl, 0, (int)(end)*sizeof(float));
	cumpnl=(double*) calloc(end+1,sizeof(double));
	//zscores=(float*) calloc(end+1,sizeof(float));
	hipHostAlloc(&zscores, (int)(end)*sizeof(float), hipHostMallocDefault);
	//memset(zscores, 0, (int)(end)*sizeof(float));

	sharpp=(double*) calloc(end+1,sizeof(double));
	sumip=(double*) calloc(end+1,sizeof(double));
	varip=(double*) calloc(end+1,sizeof(double));
	stdevip=(double*) calloc(end+1,sizeof(double));
	m_aveip=(double*) calloc(end+1,sizeof(double));
	sharp=(double*) calloc(end+1,sizeof(double));
	sumi=(double*) calloc(end+1,sizeof(double));
	vari=(double*) calloc(end+1,sizeof(double));
	stdevi=(double*) calloc(end+1,sizeof(double));
	m_avei=(double*) calloc(end+1,sizeof(double));
	sumv=(double*) calloc(end+1,sizeof(double));
	varv=(double*) calloc(end+1,sizeof(double));
	stdevv=(double*) calloc(end+1,sizeof(double));
	m_avev=(double*) calloc(end+1,sizeof(double));

	//** ALLOCATE SPACE FOR MEMORY FOR CUDA-RELATED DEVICE VARIABLES**

	hipMalloc((void**)&zscores_d, (int)(end)*sizeof(float));
	//hipMemset(zscores_d, 0, (int)(end)*sizeof(float));
	hipMalloc((void**)&pos_d, (int)(end)*sizeof(float));
	hipMemset(pos_d, 0, (int)(end)*sizeof(float));
	hipMalloc((void**)&pnl_d, (int)(end)*sizeof(float));
	hipMemset(pnl_d, 0, (int)(end)*sizeof(float));
	hipMalloc((void**)&rets_d, (int)(end)*sizeof(float));

	
	starto = LOOKBACK + (STEP*(ii-1));  
	stopf = LOOKBACK + (STEP*(ii-0));   
	if(ii>1) tachy = 1;			//use to go back n peroids for max sharpe offset
	if(stopf>=end) stopf = end;

	i=0;
	while(fgets(line,100,fin)>0){
		sscanf(line,"%ld %lf %lf %lf %lf",&dt[i],&op[i],&hi[i],&lo[i],&p[i]);i++;}  //scan lines from data file and store in arrays - this is price data here//
	f_close(source,fin);															//close data file
	ret(p, end, rets);
	lensz = (int)(sharplist[gg][2][ii-tachy] );
	zscore(lensz, p, sumv, varv, end, zscores, stdevv, m_avev);
	
	//** COPY CUDA VARIABLES FROM CPU (HOST) TO GPU (DEVICE) - USE ASYNC TRANSFER FOR MORE SPEED SO CPU DOES NOT HAVE TO WAIT FOR GPU TO FINISH OPERATION AND CAN PROCEED FURTHER IN THE MAIN PROGRAM**
	hipMemcpyAsync(zscores_d, zscores, (int)(end)*sizeof(float), hipMemcpyHostToDevice,0);
	hipMemcpyAsync(rets_d, rets, (int)(end)*sizeof(float), hipMemcpyHostToDevice,0);

	lenny=stopf-starto;
	dim3 threads; threads.x = 896;  //use 896 threads as per specific GPU device for higher OCCUPANCY/USE OF CARD - trial-and-error via PROFILING
    //dim3 blocks; blocks.x = ((int)(end)/threads.x) + 1;
	//kernelSim<<<threads,blocks>>>(zscores_d,rets_d,pnl_d,pos_d,start,stop,a[z]);

	//** CALL GPU FUNCTION/KERNEL HERE FOR MODEL PARAMETER SWEEP TO GENERATE IN_SAMPLE RESULTS**
    kernelSim<<<threads,112>>>(zscores_d,rets_d,pnl_d,pos_d,starto,stopf,(float)(sharplist[gg][1][ii-tachy]),lensz);
	
	//** COPY CUDA VARIABLES/RESULTS FROM GPU (DEVICE) BACK TO CPU (HOST) - MUST WAIT FOR GPU OPERATION/FUNCTION TO FINISH HERE SINCE LOW ASYNC/CONCURRENCY ON NON_TESLA GPU DEVICES**
	hipMemcpy(pos, pos_d, (int)(end)*sizeof(float)/*stop-start*/, hipMemcpyDeviceToHost);
	hipMemcpy(pnl, pnl_d, (int)(end)*sizeof(float), hipMemcpyDeviceToHost);
	

	//for(i=starto;i<stopf;i++){														//OUT-OF-SAMPLE runs for old CPU CODE - NOW WE'RE USING GPU INSTEAD FOR MORE SPEED**
	//	
	//	if(zscores[i] > sharplist[gg][1][ii-tachy]) pos[i] = 1.00;														
	//	if(zscores[i] < -sharplist[gg][1][ii-tachy]) pos[i] = -1.00;	
	//	pnl[i] = (pos[i] * rets[i]);}
		
		
		
		
		sharpep(pnl, sumip, varip, starto, stopf, stdevip, m_aveip, sharpp);	
		//ocum[gg][0][ii] = cumpnl[stopf-1];
		//ocum[gg][1][ii] = cumdolls[stopf-1];
		ocum[gg][2][ii] = sharpp[stopf-1];
		if (sharpp[stopf-1] > 0.00) combos = combos + 1; 
		sharpies = sharpies + sharpp[stopf-1]/(peri[gg]);
		counters=counters+1;
		mat =(((((ii+0)*(double)(STEP)))+LOOKBACK)/260);
		if (mat >= ((double)(end)/260)) mat = ((double)(end)/260);
		if (stop>=end) mat = ((double)(end)/260);
		printf("\nOS PNL: Market%d-%s Period%d", gg,marks[gg-1],ii);		
		printf("\nOS-Sharpe\t%.2f", sharpp[stopf-1]);
		printf("\nOS-Param1\t%.2f", sharplist[gg][1][ii-tachy]);
		printf("\nOS-Param2\t%.0f", sharplist[gg][2][ii-tachy]);
		//printf("\nOS-Cum Ret\t%.2f%%", cumpnl[stopf-1]*100);
		//printf("\nOS-Ann Ret\t%.2f%%", (cumpnl[stopf-1]*100)/(LOOKBACK/260));
		//printf("\nOS-Ann Vol\t%.2f%%", ABS(((cumpnl[stopf-1]*100)/(LOOKBACK/260))/sharpp[stopf-1]));
		printf("\nNum of Years: %.2f thru %.2f of %.2f total\n", (((((ii-1)*(double)(STEP)))+LOOKBACK)/260),mat,((double)(end)/260));
		fprintf(recon,"\nOS PNL: Market%d-%s Period%d", gg,marks[gg-1],ii);		
		fprintf(recon,"\nOS-Sharpe\t%.2f", sharpp[stopf-1]);
		fprintf(recon,"\nOS-Param1\t%.2f", sharplist[gg][1][ii-tachy]);
		fprintf(recon,"\nOS-Param2\t%.0f", sharplist[gg][2][ii-tachy]);
		//fprintf(recon,"\nOS-Cum Ret\t%.2f%%", cumpnl[stopf-1]*100);
		//fprintf(recon,"\nOS-Ann Ret\t%.2f%%", (cumpnl[stopf-1]*100)/(LOOKBACK/260));
		//fprintf(recon,"\nOS-Ann Vol\t%.2f%%", ABS(((cumpnl[stopf-1]*100)/(LOOKBACK/260))/sharpp[stopf-1]));
		fprintf(recon,"\nNum of Years: %.2f thru %.2f of %.2f total\n", (((((ii-1)*(double)(STEP)))+LOOKBACK)/260),mat,((double)(end)/260));
		
	for(i=starto;i<stopf;i++) {
		fprintf(ferr,"%ld\t %10.6lf\t %10.3lf\t %10.2lf\t %10.5lf\t %10.5lf\t %10.5lf\n",dt[i],p[i],zscores[i],pos[i],rets[i],pnl[i],sharpp[i]);
		fprintf(ferri,"%ld\t %10.6lf\t %10.3lf\t %10.2lf\t %10.5lf\t %10.5lf\t %10.5lf\n",dt[i],p[i],zscores[i],pos[i],rets[i],pnl[i],sharpp[i]+ocum[gg][2][ii-1]); } //	

	
	f_close(dest,ferr);							//close output file

	if (sharpp[stopf-1] > 0.00) winos = winos+ii/(peri[gg]);
	/*ocum[gg][0][ii] = cumpnl[stopf-1];
	ocum[gg][1][ii] = cumdolls[stopf-1];
	ocum[gg][2][ii] = sharpp[stopf-1];*/
	tachy=0;

	free(hi);free(lo);free(p);free(price);free(smooth);free(detrender);free(period);free(qu);free(iu);free(ji);free(jq);free(ib);free(qb);free(sib);free(re);free(im);free(sre);free(sim);free(speriod);free(smperiod);free(qc);free(ic);free(ric);free(intperiod);free(sig);free(nois);free(snr);
	hipHostFree(rets);hipHostFree(zscores);free(cumpnl);free(op);free(sharp);free(sumi);free(vari);free(stdevi);free(m_avei);//hipFree(zscores);hipFree(rets);hipFree(pnl);hipFree(pos);
	free(sumv);free(varv);free(stdevv);free(m_avev);free(dolls);free(cumdolls);free(sharpp);free(sumip);free(varip);free(stdevip);free(m_aveip);hipHostFree(pos);hipHostFree(pnl);
	hipFree(zscores_d);hipFree(pnl_d);hipFree(pos_d);hipFree(rets_d);

	}
	
	f_close(desta,ferri);						//close output file

}
	
	t2 = clock();   
	diff = (((float)t2 - (float)t1) / 1000000.0F ) * 1000;   
	printf("\n\n%.2f min..Avg I-MaxSharpe is %.2f..+I-Sharpes are %d of %d combos or %.0f%%..\n+%d Win Markets out of %i..Avg All I-Sharpes is %.2f\n\n",diff/60.0,avv/NUMI,combo,counter,(((double)(combo)/(double)(counter))*100),wins,NUMI,sharpie/(double)(counter)); 
	fprintf(recon,"\n\n%.2f min..Avg I-MaxSharpe is %.2f..+I-Sharpes are %d of %d combos or %.0f%%..\n+%d Win Markets out of %i..Avg All I-Sharpes is %.2f\n\n",diff/60.0,avv/NUMI,combo,counter,(((double)(combo)/(double)(counter))*100),wins,NUMI,sharpie/(double)(counter)); 
	
	printf("\n\n%.2f min..+O-Sharpes are %d of %d combos or %.0f%%..\n+%d Win Markets out of %i..Avg All O-Sharpes is %.2f\n\n",diff/60.0,combos,counters,(((double)(combos)/(double)(counters))*100),winos,NUMI,sharpies/(double)(NUMI));//(double)(counters)); //NUMI
	fprintf(recon,"\n\n%.2f sec..+O-Sharpes are %d of %d combos or %.0f%%..\n+%d Win Markets out of %i..Avg All O-Sharpes is %.2f\n\n",diff/60.0,combos,counters,(((double)(combos)/(double)(counters))*100),winos,NUMI,sharpies/(double)(NUMI)); //NUMI
	
	f_close(PATH"recon.dat",recon); 
	
	//**RESET GPU DEVICE**//
	hipDeviceReset();
	system("pause");
	
}




















